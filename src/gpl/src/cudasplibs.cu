#include "hip/hip_runtime.h"
#include "cudasplibs.h"

void cudaerror(hipError_t code) {
    if (code != hipSuccess){
        fprintf(stderr, "Error %s at line %d in file %s\n", hipGetErrorString(code), __LINE__, __FILE__); 
        exit(-1);
    }
}
void cusparseerror(hipsparseStatus_t code) {
    if (code != HIPSPARSE_STATUS_SUCCESS){
        fprintf(stderr, "Error %d at line %d in file %s\n", int(code), __LINE__, __FILE__); 
        exit(-1);
    }
}

void cusolvererror(hipsolverStatus_t code) {
    if (code != HIPSOLVER_STATUS_SUCCESS){
        fprintf(stderr, "Error %d at line %d in file %s\n", int(code), __LINE__, __FILE__); 
        exit(-1);
    }
}

cudasplibs::cudasplibs(std::vector<int>& cooRowIndex, std::vector<int>& cooColIndex, std::vector<float>& cooVal, Eigen::VectorXf& fixedInstForceVec){
    m = fixedInstForceVec.size();
    nnz = cooVal.size();

    // Allocate device memeory and copy data to device
    cudaerror(hipMalloc((void**)&d_cooRowIndex, nnz * sizeof(int)));
    cudaerror(hipMalloc((void**)&d_cooColIndex, nnz * sizeof(int)));
    cudaerror(hipMalloc((void**)&d_cooVal, nnz * sizeof(float)));
    cudaerror(hipMalloc((void**)&d_fixedInstForceVec, m * sizeof(float)));
    cudaerror(hipMalloc((void**)&d_instLocVec, m * sizeof(float)));
          
    // Copy data (COO storage method)
    cudaerror(hipMemcpy(d_cooRowIndex, cooRowIndex.data(), sizeof(int)*nnz, hipMemcpyHostToDevice));
    cudaerror(hipMemcpy(d_cooColIndex, cooColIndex.data(), sizeof(int)*nnz, hipMemcpyHostToDevice));
    cudaerror(hipMemcpy(d_cooVal, cooVal.data(), sizeof(float)*nnz, hipMemcpyHostToDevice));
    cudaerror(hipMemcpy(d_fixedInstForceVec, fixedInstForceVec.data(), sizeof(float)*m, hipMemcpyHostToDevice));
}

void cudasplibs::cusolverSpQR(Eigen::VectorXf& instLocVec){
    // Parameters that don't change with iteration and used in the CUDA code
    float tol = 1e-6; // 	Tolerance to decide if singular or not.
    int reorder = 0;  // "0" for common matrix without ordering
    int singularity = -1; // Output. -1 = A means invertible
    
    // Set handler
    hipsolverSpHandle_t handleCusolver = NULL;
    hipsparseHandle_t handleCusparse = NULL;
    hipStream_t streamX = NULL;
    
    // Initialize handler
    cusolvererror(hipsolverSpCreate(&handleCusolver));
    cusparseerror(hipsparseCreate(&handleCusparse));
    cudaerror(hipStreamCreate(&streamX));
    cusolvererror(hipsolverSpSetStream(handleCusolver, streamX));
    cusparseerror(hipsparseSetStream(handleCusparse, streamX));

    // Create and define cusparse descriptor
    hipsparseMatDescr_t descr = NULL;
    cusparseerror(hipsparseCreateMatDescr(&descr));
    cusparseerror(hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
    cusparseerror(hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO));

    // transform from coordinates (COO) values to compressed row pointers (CSR) values
    // https://docs.nvidia.com/cuda/cusparse/index.html
    int *d_csrRowInd = NULL;
    cudaerror(hipMalloc((void**)&d_csrRowInd, (m+1) * sizeof(int)));
    cusparseerror(hipsparseXcoo2csr(handleCusparse, d_cooRowIndex, nnz, m, d_csrRowInd, HIPSPARSE_INDEX_BASE_ZERO));
    cusolvererror(hipsolverSpScsrlsvqr(handleCusolver, m, nnz, descr, d_cooVal, d_csrRowInd, d_cooColIndex, d_fixedInstForceVec, tol, reorder, d_instLocVec, &singularity));

    // Sync and Copy data to host
    cudaerror(hipMemcpyAsync(instLocVec.data(), d_instLocVec, sizeof(float)*m, hipMemcpyDeviceToHost, streamX));

    cudaerror(hipFree(d_csrRowInd));
    cusparseerror(hipsparseDestroyMatDescr( descr ) );
    cusparseerror(hipsparseDestroy(handleCusparse));
    cusolvererror(hipsolverSpDestroy(handleCusolver));  
}

__global__ void Multi_MatVec(float error, int nnz, int m , float* d_Ax, float* d_fixedInstForceVec, float* d_instLocVec, int* d_cooRowIndex, int* d_cooColIndex, float* d_cooVal){
    float sum = 0;
    int num = blockIdx.x * blockDim.x + threadIdx. x;
    if (num < nnz){
        d_Ax[d_cooRowIndex[num]] += d_cooVal[num] * d_instLocVec[d_cooColIndex[num]];
    }
    for (size_t row = 0; row < m; row++){
        sum += (d_fixedInstForceVec[row] > 0) ? d_fixedInstForceVec[row] : -d_fixedInstForceVec[row];
        if (d_fixedInstForceVec[row] > d_Ax[row])
            error += d_fixedInstForceVec[row] - d_Ax[row];
        else
            error -= d_fixedInstForceVec[row]- d_Ax[row];
    }
    if (sum != 0) error = error / sum;
}

float cudasplibs::error_cal(){
    float error = 0;
    float *d_Ax;
    cudaerror(hipMalloc((void**)&d_Ax, m * sizeof(float)));
    cudaerror(hipMemset(d_Ax, 0, m));
    unsigned int threads = 512;
    unsigned int blocks = (m+threads-1) / threads;
    Multi_MatVec<<<blocks, threads>>>(error, nnz, m, d_Ax, d_fixedInstForceVec, d_instLocVec, d_cooRowIndex, d_cooColIndex, d_cooVal);
    cudaerror(hipFree(d_Ax));
    return (error > 0) ? error : -error;
}

void cudasplibs::release(){
          
    // Destroy what is not needed in both of device and host
    cudaerror(hipFree(d_cooColIndex));
    cudaerror(hipFree(d_cooRowIndex));
    cudaerror(hipFree(d_cooVal));
    cudaerror(hipFree(d_instLocVec));
    cudaerror(hipFree(d_fixedInstForceVec));

}