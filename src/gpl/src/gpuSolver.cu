#include "hip/hip_runtime.h"


#include "gpuSolver.h"

namespace gpl {
using namespace std;
using utl::GPL;

void GpuSolver::cudaerror(hipError_t code)
{
  if (code != hipSuccess) {
    log_->error(GPL, 1, "[CUDA ERROR] {} at line {} in file {} \n",
                 hipGetErrorString(code),
                 __LINE__,
                 __FILE__);
    hipDeviceReset();
  }
}
void GpuSolver::cusparseerror(hipsparseStatus_t code)
{
  if (code != HIPSPARSE_STATUS_SUCCESS) {
    log_->error(GPL, 1, "[CUSPARSE ERROR] {} at line {} in file {}\n",
                 hipsparseGetErrorString(code),
                 __LINE__,
                 __FILE__);
    hipDeviceReset();
  }
}

void GpuSolver::cusolvererror(hipsolverStatus_t code)
{
  if (code != HIPSOLVER_STATUS_SUCCESS) {
    log_->error(GPL, 1, "[CUSOLVER ERROR] {} at line {} in file {}\n",
                 hipGetErrorString(*(hipError_t*) &code),
                 __LINE__,
                 __FILE__);
    hipDeviceReset();
  }
}

GpuSolver::GpuSolver(){}

GpuSolver::GpuSolver(SMatrix& placeInstForceMatrix, Eigen::VectorXf& fixedInstForceVec, utl::Logger* logger)
{
    // {cooRowIndex_, cooColIndex_, cooVal_} is the triplet vector stored as COO format to represent placeInstForceMatrix
    vector<int> cooRowIndex_, cooColIndex_;
    vector<float> cooVal_;

    for (size_t row = 0; row < placeInstForceMatrix.rows(); row++) {
      for (size_t col = 0; col < placeInstForceMatrix.cols(); col++) {
        if (placeInstForceMatrix.coeffRef(row, col) != 0) {
          cooRowIndex_.push_back(row);
          cooColIndex_.push_back(col);
          cooVal_.push_back(placeInstForceMatrix.coeffRef(row, col));
        }
      }
    }

    m_ = fixedInstForceVec.size();
    nnz_ = cooVal_.size();
    log_ = logger;

    cudaerror(hipMalloc((void**)&d_cooRowIndex_, nnz_ * sizeof(int)));
    cudaerror(hipMalloc((void**)&d_cooColIndex_, nnz_ * sizeof(int)));
    cudaerror(hipMalloc((void**)&d_cooVal_, nnz_ * sizeof(float)));
    cudaerror(hipMalloc((void**) &d_fixedInstForceVec_, m_ * sizeof(float)));
    cudaerror(hipMalloc((void**) &d_instLocVec_, m_ * sizeof(float)));

    //Copy data (COO storage method)
    cudaerror(hipMemcpy(d_cooRowIndex_, cooRowIndex_.data(), sizeof(int)*nnz_,
    hipMemcpyHostToDevice));
    cudaerror(hipMemcpy(d_cooColIndex_,
                        cooColIndex_.data(),
                        sizeof(int) * nnz_,
                        hipMemcpyHostToDevice));
    cudaerror(hipMemcpy(d_cooVal_, cooVal_.data(), sizeof(float)*nnz_,
    hipMemcpyHostToDevice));
    cudaerror(hipMemcpy(d_fixedInstForceVec_,
                        fixedInstForceVec.data(),
                        sizeof(float) * m_,
                        hipMemcpyHostToDevice));

    std:: cout << "Yes!" << std::endl;
}

void GpuSolver::cusolverCal(Eigen::VectorXf& instLocVec){
  // Parameters that don't change with iteration and used in the CUDA code
  float tol = 1e-6;      // 	Tolerance to decide if singular or not.
  int reorder = 0;       // "0" for common matrix without ordering
  int singularity = -1;  // Output. -1 = A means invertible

  // Set handler
  hipsolverSpHandle_t handleCusolver = NULL;
  hipsparseHandle_t handleCusparse = NULL;
  hipStream_t stream = NULL;

  // Initialize handler
  cusolvererror(hipsolverSpCreate(&handleCusolver));
  cusparseerror(hipsparseCreate(&handleCusparse));
  cudaerror(hipStreamCreate(&stream));
  cusolvererror(hipsolverSpSetStream(handleCusolver, stream));
  cusparseerror(hipsparseSetStream(handleCusparse, stream));

  // Create and define cusparse descriptor
  hipsparseMatDescr_t descr = NULL;
  cusparseerror(hipsparseCreateMatDescr(&descr));
  cusparseerror(hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
  cusparseerror(hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO));

  // transform from coordinates (COO) values to compressed row pointers (CSR)
  // values https://docs.nvidia.com/cuda/cusparse/index.html
  int* d_csrRowInd = NULL;
  //   thrust::device_vector<int> t_csrRowInd(m_ + 1, 0);
  // d_csrRowInd = thrust::raw_pointer_cast(t_csrRowInd.data());
  cudaerror(hipMalloc((void**)&d_csrRowInd, (m_+1) * sizeof(int)));
  cusparseerror(hipsparseXcoo2csr(handleCusparse,
                                 d_cooRowIndex_,
                                 nnz_,
                                 m_,
                                 d_csrRowInd,
                                 HIPSPARSE_INDEX_BASE_ZERO));

  cusolvererror(hipsolverSpScsrlsvqr(handleCusolver,
                                    m_,
                                    nnz_,
                                    descr,
                                    d_cooVal_,
                                    d_csrRowInd,
                                    d_cooColIndex_,
                                    d_fixedInstForceVec_,
                                    tol,
                                    reorder,
                                    d_instLocVec_,
                                    &singularity));

  // Sync and Copy data to host
  cudaerror(hipMemcpyAsync(instLocVec.data(),
                            d_instLocVec_,
                            sizeof(float) * m_,
                            hipMemcpyDeviceToHost,
                            stream));

  cudaerror(hipFree(d_csrRowInd));
  cusparseerror(hipsparseDestroyMatDescr( descr ) );
  cusparseerror(hipsparseDestroy(handleCusparse));
  cusolvererror(hipsolverSpDestroy(handleCusolver));
}

__global__ void Multi_MatVec(float error,
                             int nnz_,
                             int m_,
                             float* d_Ax,
                             float* d_fixedInstForceVec_,
                             float* d_instLocVec_,
                             int* d_cooRowIndex_,
                             int* d_cooColIndex_,
                             float* d_cooVal_)
{
  float sum = 0;
  int num = blockIdx.x * blockDim.x + threadIdx.x;
  if (num < nnz_) {
    d_Ax[d_cooRowIndex_[num]]
        += d_cooVal_[num] * d_instLocVec_[d_cooColIndex_[num]];
  }
  for (size_t row = 0; row < m_; row++) {
    sum += (d_fixedInstForceVec_[row] > 0) ? d_fixedInstForceVec_[row]
                                           : -d_fixedInstForceVec_[row];
    if (d_fixedInstForceVec_[row] > d_Ax[row])
      error += d_fixedInstForceVec_[row] - d_Ax[row];
    else
      error -= d_fixedInstForceVec_[row] - d_Ax[row];
  }
  if (sum != 0)
    error = error / sum;
}

float GpuSolver::error_cal()
{
  float error = 0;
  float* d_Ax;
  cudaerror(hipMalloc((void**)&d_Ax, m_ * sizeof(float)));
  cudaerror(hipMemset(d_Ax, 0, m_));
  unsigned int threads = 512;
  unsigned int blocks = (m_ + threads - 1) / threads;
  Multi_MatVec<<<blocks, threads>>>(error,
                                    nnz_,
                                    m_,
                                    d_Ax,
                                    d_fixedInstForceVec_,
                                    d_instLocVec_,
                                    d_cooRowIndex_,
                                    d_cooColIndex_,
                                    d_cooVal_);
  cudaerror(hipFree(d_Ax));
  return (error > 0) ? error : -error;
}

GpuSolver::~GpuSolver()
{
  // Destroy what is not needed in both of device and host
  cudaerror(hipFree(d_cooColIndex_));
  cudaerror(hipFree(d_cooRowIndex_));
  cudaerror(hipFree(d_cooVal_));
  cudaerror(hipFree(d_instLocVec_));
  cudaerror(hipFree(d_fixedInstForceVec_));
}
}
