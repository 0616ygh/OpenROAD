#include "hip/hip_runtime.h"
#include "gpuSolver.h"

namespace gpl {
using namespace std;
using utl::GPL;

void GpuSolver::cudaerror(hipError_t code)
{
  if (code != hipSuccess) {
    log_->error(GPL,
                1,
                "[CUDA ERROR] {} at line {} in file {} \n",
                hipGetErrorString(code),
                __LINE__,
                __FILE__);
  }
}
void GpuSolver::cusparseerror(hipsparseStatus_t code)
{
  if (code != HIPSPARSE_STATUS_SUCCESS) {
    log_->error(GPL,
                1,
                "[CUSPARSE ERROR] {} at line {} in file {}\n",
                hipsparseGetErrorString(code),
                __LINE__,
                __FILE__);
  }
}

void GpuSolver::cusolvererror(hipsolverStatus_t code)
{
  if (code != HIPSOLVER_STATUS_SUCCESS) {
    log_->error(GPL,
                1,
                "[CUSOLVER ERROR] {} at line {} in file {}\n",
                hipGetErrorString(*(hipError_t*) &code),
                __LINE__,
                __FILE__);
  }
}

GpuSolver::GpuSolver()
{
}

GpuSolver::GpuSolver(SMatrix& placeInstForceMatrix,
                     Eigen::VectorXf& fixedInstForceVec,
                     utl::Logger* logger)
{
  // {cooRowIndex_, cooColIndex_, cooVal_} are the host vectors used to store
  // the sparse format of placeInstForceMatrix.
  nnz_ = placeInstForceMatrix.nonZeros();
  vector<int> cooRowIndex, cooColIndex;
  vector<float> cooVal;
  cooRowIndex.reserve(nnz_);
  cooColIndex.reserve(nnz_);
  cooVal.reserve(nnz_);

  for(int row = 0; row < placeInstForceMatrix.outerSize(); row++){
    for(typename Eigen::SparseMatrix<float,Eigen::RowMajor>::InnerIterator it(placeInstForceMatrix,row); it; ++it){
      cooRowIndex.push_back(it.row());
      cooColIndex.push_back(it.col());
      cooVal.push_back(it.value());
    }
  }

  m_ = fixedInstForceVec.size();
  nnz_ = cooVal.size();
  log_ = logger;
  d_cooRowIndex_.resize(nnz_);
  d_cooColIndex_.resize(nnz_);
  d_cooVal_.resize(nnz_);
  d_fixedInstForceVec_.resize(m_);
  d_instLocVec_.resize(m_);

  // Copy the COO formatted triplets to device
  thrust::copy(cooRowIndex.begin(), cooRowIndex.end(), d_cooRowIndex_.begin());
  thrust::copy(cooColIndex.begin(), cooColIndex.end(), d_cooColIndex_.begin());
  thrust::copy(cooVal.begin(), cooVal.end(), d_cooVal_.begin());
  thrust::copy(&fixedInstForceVec[0],
               &fixedInstForceVec[m_ - 1],
               d_fixedInstForceVec_.begin());

  // Set raw pointers to point to the triplets in the device
  r_cooRowIndex_ = thrust::raw_pointer_cast(d_cooRowIndex_.data());
  r_cooColIndex_ = thrust::raw_pointer_cast(d_cooColIndex_.data());
  r_cooVal_ = thrust::raw_pointer_cast(d_cooVal_.data());
  r_fixedInstForceVec_ = thrust::raw_pointer_cast(d_fixedInstForceVec_.data());
  r_instLocVec_ = thrust::raw_pointer_cast(d_instLocVec_.data());
}

void GpuSolver::cusolverCal(Eigen::VectorXf& instLocVec)
{
  // Parameters that don't change with iteration and used in the CUDA code
  const float tol = 1e-6;      // 	Tolerance to decide if singular or not.
  const int reorder = 0;       // "0" for common matrix without ordering
  int singularity = -1;  // Output. -1 = A means invertible

  // Set handler
  hipsolverSpHandle_t handleCusolver = NULL;
  hipsparseHandle_t handleCusparse = NULL;
  hipStream_t stream = NULL;

  // Initialize handler
  cusolvererror(hipsolverSpCreate(&handleCusolver));
  cusparseerror(hipsparseCreate(&handleCusparse));
  cudaerror(hipStreamCreate(&stream));
  cusolvererror(hipsolverSpSetStream(handleCusolver, stream));
  cusparseerror(hipsparseSetStream(handleCusparse, stream));

  // Create and define cusparse descriptor
  hipsparseMatDescr_t descr = NULL;
  cusparseerror(hipsparseCreateMatDescr(&descr));
  cusparseerror(hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
  cusparseerror(hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO));

  // transform from coordinates (COO) values to compressed row pointers (CSR)
  // values https://docs.nvidia.com/cuda/cusparse/index.html
  int* r_csrRowInd = NULL;
  thrust::device_vector<int> d_csrRowInd(m_ + 1, 0);
  r_csrRowInd = thrust::raw_pointer_cast(d_csrRowInd.data());

  cusparseerror(hipsparseXcoo2csr(handleCusparse,
                                 r_cooRowIndex_,
                                 nnz_,
                                 m_,
                                 r_csrRowInd,
                                 HIPSPARSE_INDEX_BASE_ZERO));

  cusolvererror(hipsolverSpScsrlsvqr(handleCusolver,
                                    m_,
                                    nnz_,
                                    descr,
                                    r_cooVal_,
                                    r_csrRowInd,
                                    r_cooColIndex_,
                                    r_fixedInstForceVec_,
                                    tol,
                                    reorder,
                                    r_instLocVec_,
                                    &singularity));

  // Sync and Copy data to host
  cudaerror(hipMemcpyAsync(instLocVec.data(),
                            r_instLocVec_,
                            sizeof(float) * m_,
                            hipMemcpyDeviceToHost,
                            stream));

  // cudaerror(hipFree(r_csrRowInd));
  cusparseerror(hipsparseDestroyMatDescr(descr));
  cusparseerror(hipsparseDestroy(handleCusparse));
  cusolvererror(hipsolverSpDestroy(handleCusolver));
}

__global__ void Multi_MatVec(float error,
                             int nnz_,
                             int m_,
                             float* r_Ax,
                             float* r_fixedInstForceVec_,
                             float* r_instLocVec_,
                             int* r_cooRowIndex_,
                             int* r_cooColIndex_,
                             float* r_cooVal_)
{
  float sum = 0;
  int num = blockIdx.x * blockDim.x + threadIdx.x;
  if (num < nnz_) {
    r_Ax[r_cooRowIndex_[num]]
        += r_cooVal_[num] * r_instLocVec_[r_cooColIndex_[num]];
  }
  for (size_t row = 0; row < m_; row++) {
    sum += (r_fixedInstForceVec_[row] > 0) ? r_fixedInstForceVec_[row]
                                           : -r_fixedInstForceVec_[row];
    if (r_fixedInstForceVec_[row] > r_Ax[row])
      error += r_fixedInstForceVec_[row] - r_Ax[row];
    else
      error -= r_fixedInstForceVec_[row] - r_Ax[row];
  }
  if (sum != 0)
    error = error / sum;
}

float GpuSolver::error()
{
  float* r_Ax;
  thrust::device_vector<float> t_Ax(m_);
  thrust::fill(t_Ax.begin(), t_Ax.end(), 0);
  r_Ax = thrust::raw_pointer_cast(t_Ax.data());

  unsigned int threads = 512;
  unsigned int blocks = (m_ + threads - 1) / threads;
  Multi_MatVec<<<blocks, threads>>>(error_,
                                    nnz_,
                                    m_,
                                    r_Ax,
                                    r_fixedInstForceVec_,
                                    r_instLocVec_,
                                    r_cooRowIndex_,
                                    r_cooColIndex_,
                                    r_cooVal_);
  return (error_ > 0) ? error_ : -error_;
}

GpuSolver::~GpuSolver()
{
}
}  // namespace gpl
